#include "hip/hip_runtime.h"
/* gib_cuda_checksum.cu: CUDA kernels for Reed-Solomon coding.
 *
 * Copyright (C) University of Alabama at Birmingham and Sandia
 * National Laboratories, 2010, written by Matthew L. Curry
 * <mlcurry@sandia.gov>
 *
 * Changes:
 * - Converted to GGEMM implementation.
 */

typedef unsigned char byte;
__device__ unsigned char gf_log_d[256];
__device__ unsigned char gf_ilog_d[256];
__constant__ byte F_d[M*N];
__constant__ byte inv_d[N*N];

/* The "fetch" datatype is the unit for performing data copies between areas of
 * memory on the GPU.  While today's wisdom says that 32-bit types are optimal
 * for this, I want to easily experiment with the others.
 */
typedef int fetch;
#define nthreadsPerBlock 128

/* These quantities must be hand-recalculated, as the compiler doesn't seem to
 * always do such things at compile time.
 */
/* fetchsize = nthreadsPerBlock * sizeof(fetch) */
#define fetchsize 512
/* size of fetch, i.e. sizeof(fetch)*/
#define SOF 4
#define nbytesPerThread SOF

#define ROUNDUPDIV(x,y) ((x + y - 1) / y)

/* We're pulling buffers from main memory based on the fetch type, but want
 * to index into it at the byte level.
 */
union shmem_bytes {
  fetch f;
  byte b[SOF];
};

/* Shared memory copies of pertinent data */
__shared__ byte sh_log[256];
__shared__ byte sh_ilog[256];

__device__ __inline__ void load_tables(uint3 threadIdx, const dim3 blockDim) {
  /* Fully arbitrary routine for any blocksize and fetch size to load
   * the log and ilog tables into shared memory.
   */
	int iters = ROUNDUPDIV(256, fetchsize);
	for (int i = 0; i < iters; i++) {
		if (i * fetchsize / SOF + threadIdx.x < 256 / SOF) {
			int fetchit = threadIdx.x + i*fetchsize/SOF;
			((fetch *)sh_log)[fetchit] =
				*(fetch *)(&gf_log_d[fetchit*SOF]);
			((fetch *)sh_ilog)[fetchit] =
				*(fetch *)(&gf_ilog_d[fetchit*SOF]);
		}
	}
}

__device__ void gib_ggemm_d(int m, int n, int k,
			    shmem_bytes *A, int lda,
			    byte *B, int ldb,
			    unsigned char beta, shmem_bytes *C, int ldc) {
	/* Previous parameters: shmem_bytes *in_bufs, int buf_size) */
	/* Requirement:
	   lda % SOF == 0.  This prevents expensive divide operations.
	   Notes:
           - The 'm' parameter is explicit in the call. Unless this
             kernel is to check bounds on m, passing this parameter is
             not necessary.
	   - B is considered to be efficiently accessed in-place; no
             loading into faster areas is attempted. Access is not
             coordinated.
	   - This kernel is optimised for large m and small n, k.
	   - Compile time parameters: M (max n) and N (max k). Here,
             only M is used.
	   - There is a bug affecting CUDA compilers from version 2.3
	     onward that causes this kernel to miscompile for M=2. For
	     this case, there is some preprocessor trickiness that
	     allows this kernel to execute M=3, but only store for
	     M=2.
	*/

#if M == 2
#undef M
#define M 3
#define RAID6_FIX
#endif

	int rank = threadIdx.x + __umul24(blockIdx.x, blockDim.x);
	load_tables(threadIdx, blockDim);
	__syncthreads();

	shmem_bytes out[M];
	shmem_bytes in;

	if (beta == 0) {
		for (int i = 0; i < M; i++) {
			out[i].f = 0;
		}
	} else {
		for (int i = 0; i < n; i++) {
			out[i].f = C[rank + lda / SOF * i].f;
			int b_tmp = sh_log[beta];
			for (int b = 0; b < SOF; b++) {
				int sum = b_tmp + sh_log[(out[i].b)[b]];
				if (sum >= 255) sum -= 255;
				(out[i].b)[b] = sh_ilog[sum];
			}
		}
	}

	for (int i = 0; i < k; ++i) {
		in.f = A[rank + lda / SOF * i].f;
		for (int j = 0; j < n; ++j) {
			/* If I'm not hallucinating, this conditional really
			   helps on the 8800 stuff, but it hurts on the 260.
			*/
			//if (F_d[j*k+i] != 0) {
			int F_tmp = sh_log[B[j * ldb + i]];
			for (int b = 0; b < SOF; ++b) {
				if (in.b[b] != 0) {
					int sum_log = F_tmp +
						sh_log[(in.b)[b]];
					if (sum_log >= 255) sum_log -= 255;
					(out[j].b)[b] ^= sh_ilog[sum_log];
				}
			}
			//}
		}
	}

#ifdef RAID6_FIX
#undef M
#define M 2
#undef RAID6_FIX
#endif
	/* This works as long as buf_size % blocksize == 0 */
	for (int i = 0; i < n/*M*/; i++)
		C[rank + ldc / SOF * i].f = out[i].f;
}

__global__ void gib_generate_d(shmem_bytes *bufs, int buf_size)
{
	gib_ggemm_d(buf_size, M, N,
		    bufs, buf_size,
		    F_d, N,
		    0, bufs + buf_size * N / SOF, buf_size);
}

__global__ void gib_recover_d(shmem_bytes *bufs, int buf_size,
			       int recover_last)
{
	gib_ggemm_d(buf_size, recover_last, N,
		    bufs, buf_size,
		    F_d, N,
		    0, bufs + buf_size * N / SOF, buf_size);
}
